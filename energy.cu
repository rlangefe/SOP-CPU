#include "hip/hip_runtime.h"
#include <cstdlib>
#include <math.h>
#include <cstdio>
#include "global.h"
#include "energy.h"

#define SECTION_SIZE 1024

__device__ __constant__ double dev_coeff_att[3][3] = { 
  {0.0, 0.0, 0.0},
  {0.0, 0.7, 0.8},
  {0.0, 0.8, 1.0} 
};

__device__ __constant__ double dev_coeff_rep[3][3] = { 
  {0.0, 0.0, 0.0},
  {0.0, 1.0, 1.0},
  {0.0, 1.0, 1.0} 
};

__device__ __constant__ double dev_sigma_rep[3][3] = {
	{0.0, 0.0, 0.0},
	{0.0, 3.8, 5.4},
	{0.0, 5.4, 7.0}
};

__device__ __constant__ double dev_force_coeff_att[3][3] = { 
  {0.0,       0.0,       0.0},
  {0.0, -12.0*1.0, -12.0*0.8},
	{0.0, -12.0*0.8, -12.0*0.7}
};

__device__ double dev_force_coeff_rep[3][3] = {
  {0.0,       0.0,       0.0},
	{0.0,  -6.0*1.0,  -6.0*1.0},
	{0.0,  -6.0*1.0,  -6.0*1.0}
};

void energy_eval()
{

  using namespace std;
  char oline[1024];

  for( int i=1; i<=npot_term; i++ ) {
    pot_term[i]();
  }

  rna_etot = e_bnd + e_ang_ss + e_vdw_rr;
  system_etot = rna_etot;

}

void force_eval()
{

  using namespace std;
  char oline[1024];

  clear_forces();

  for( int i=1; i<=nforce_term; i++ ) {
    force_term[i]();
  }

}

void clear_forces() {

  using namespace std;

  for( int i=1; i<=nbead; i++ ) {
    force[i].x = 0.0;
    force[i].y = 0.0;
    force[i].z = 0.0;
  }

}

void set_potential() {

  using namespace std;

  int iterm;

  iterm = 0;
  for( int i=1; i<=mpot_term; i++ ) {
    switch(i) {
    case 1:
      if( pot_term_on[i] ) {
	      pot_term[++iterm] = &fene_energy;
      }
      break;
    case 2:
      if( pot_term_on[i] ) {
	      pot_term[++iterm] = &soft_sphere_angular_energy;
      }
      break;
    case 5:
      if( pot_term_on[i] ) {
        if(usegpu_vdw_energy == 0){
	        pot_term[++iterm] = &vdw_energy;
        }else{
          pot_term[++iterm] = &vdw_energy_gpu;
        }
      }
      break;
    default:
      break;
    }
  }

}

void set_forces()
{

  using namespace std;

  int iterm;

  iterm = 0;
  for( int i=1; i<=mforce_term; i++ ) {
    switch(i) {
    case 1:
      if( force_term_on[i] ) {
	      force_term[++iterm] = &random_force;
      }
      break;
    case 2:
      if( force_term_on[i] ) {
	      force_term[++iterm] = &fene_forces;
      }
      break;
    case 3:
      if( force_term_on[i] ) {
	      force_term[++iterm] = &soft_sphere_angular_forces;
      }
      break;
    case 6:
      if( force_term_on[i] ) {
        if(usegpu_vdw_energy == 0){
	        force_term[++iterm] = &vdw_forces;
        }else{
          force_term[++iterm] = &vdw_forces_gpu;
        }
      }
      break;
    default:
      break;
    }
  }

}

void fene_energy()
{

  using namespace std;

  int ibead, jbead;
  double dx, dy, dz, d,dev;
  char line[2048];

  e_bnd = 0.0;
  for( int i=1; i<=nbnd; i++ ) {

    ibead = ibead_bnd[i];
    jbead = jbead_bnd[i];

    dx = unc_pos[jbead].x-unc_pos[ibead].x;
    dy = unc_pos[jbead].y-unc_pos[ibead].y;
    dz = unc_pos[jbead].z-unc_pos[ibead].z;


    // min images

    dx -= boxl*rnd(dx/boxl);
    dy -= boxl*rnd(dy/boxl);
    dz -= boxl*rnd(dz/boxl);

    d = sqrt(dx*dx+dy*dy+dz*dz);
    dev = d-pdb_dist[i];

    e_bnd += log1p(-dev*dev/R0sq); // log1p(x) = log(1-x)

  }

  e_bnd *= -e_bnd_coeff;

  return;

}

void soft_sphere_angular_energy()
{

  using namespace std;

  e_ang_ss = 0.0;
  int ibead, kbead;
  coord r_ik;
  double d,d6;

    for( int i=1; i<=nang; i++ ) {

      ibead = ibead_ang[i];
      kbead = kbead_ang[i];

      r_ik.x = unc_pos[kbead].x - unc_pos[ibead].x;
      r_ik.y = unc_pos[kbead].y - unc_pos[ibead].y;
      r_ik.z = unc_pos[kbead].z - unc_pos[ibead].z;

      // min images

      r_ik.x -= boxl*rnd(r_ik.x/boxl);
      r_ik.y -= boxl*rnd(r_ik.y/boxl);
      r_ik.z -= boxl*rnd(r_ik.z/boxl);

      d = sqrt(r_ik.x*r_ik.x + r_ik.y*r_ik.y + r_ik.z*r_ik.z);
      d6 = pow(d,6.0);

      e_ang_ss += e_ang_ss_coeff/d6;
  }

  return;

}

void vdw_energy()
{

  using namespace std;

  int ibead,jbead;
  int itype,jtype;
  double dx,dy,dz,d,d2,d6,d12;
  char line[2048];

  e_vdw_rr = 0.0;
  e_vdw_rr_att = 0.0;
  e_vdw_rr_rep = 0.0;

  for( int i=1; i<=nil_att; i++ ) {

    ibead = ibead_pair_list_att[i];
    jbead = jbead_pair_list_att[i];
    itype = itype_pair_list_att[i];
    jtype = jtype_pair_list_att[i];

    dx = unc_pos[jbead].x - unc_pos[ibead].x;
    dy = unc_pos[jbead].y - unc_pos[ibead].y;
    dz = unc_pos[jbead].z - unc_pos[ibead].z;

    // min images

    dx -= boxl*rnd(dx/boxl);
    dy -= boxl*rnd(dy/boxl);
    dz -= boxl*rnd(dz/boxl);

    d2 = dx*dx+dy*dy+dz*dz;
    d6 = d2*d2*d2;
    d12 = d6*d6;

    e_vdw_rr_att += coeff_att[itype][jtype] * (pl_lj_nat_pdb_dist12[i]/d12)-2.0*(pl_lj_nat_pdb_dist6[i]/d6);

  }

  for( int i=1; i<=nil_rep; i++ ) {

    ibead = ibead_pair_list_rep[i];
    jbead = jbead_pair_list_rep[i];
    itype = itype_pair_list_rep[i];
    jtype = jtype_pair_list_rep[i];

    dx = unc_pos[jbead].x - unc_pos[ibead].x;
    dy = unc_pos[jbead].y - unc_pos[ibead].y;
    dz = unc_pos[jbead].z - unc_pos[ibead].z;

    // min images

    dx -= boxl*rnd(dx/boxl);
    dy -= boxl*rnd(dy/boxl);
    dz -= boxl*rnd(dz/boxl);

    d2 = dx*dx+dy*dy+dz*dz;
    d6 = d2*d2*d2;
    d12 = d6*d6;

    e_vdw_rr_rep += coeff_rep[itype][jtype] * (sigma_rep12[itype][jtype]/d12+sigma_rep6[itype][jtype]/d6);

  }

  e_vdw_rr = e_vdw_rr_att + e_vdw_rr_rep;

  return;

}

void vdw_forces()
{

  using namespace std;

  char line[2048];

  int ibead,jbead;
  int itype,jtype;
  double dx,dy,dz,d,d2,d6,d12;
  double fx,fy,fz;
  double co1;
  const static double tol = 1.0e-7;
  double rep_tol;

  for( int i=1; i<=nil_att; i++ ) {

    ibead = ibead_pair_list_att[i];
    jbead = jbead_pair_list_att[i];
    itype = itype_pair_list_att[i];
    jtype = jtype_pair_list_att[i];

    dx = unc_pos[jbead].x - unc_pos[ibead].x;
    dy = unc_pos[jbead].y - unc_pos[ibead].y;
    dz = unc_pos[jbead].z - unc_pos[ibead].z;

    // min images

    dx -= boxl*rnd(dx/boxl);
    dy -= boxl*rnd(dy/boxl);
    dz -= boxl*rnd(dz/boxl);

    d2 = dx*dx+dy*dy+dz*dz;
    
    if( d2 < tol*pl_lj_nat_pdb_dist2[i] ) continue;
    d6 = d2*d2*d2;
    d12 = d6*d6;

    co1 = force_coeff_att[itype][jtype]/d2*((pl_lj_nat_pdb_dist12[i]/d12)-(pl_lj_nat_pdb_dist6[i]/d6));

    fx = co1*dx;
    fy = co1*dy;
    fz = co1*dz;

    force[ibead].x += fx;
    force[ibead].y += fy;
    force[ibead].z += fz;

    force[jbead].x -= fx;
    force[jbead].y -= fy;
    force[jbead].z -= fz;

  }

  for( int i=1; i<=nil_rep; i++ ) {

    ibead = ibead_pair_list_rep[i];
    jbead = jbead_pair_list_rep[i];
    itype = itype_pair_list_rep[i];
    jtype = jtype_pair_list_rep[i];

    dx = unc_pos[jbead].x - unc_pos[ibead].x;
    dy = unc_pos[jbead].y - unc_pos[ibead].y;
    dz = unc_pos[jbead].z - unc_pos[ibead].z;

    // min images

    dx -= boxl*rnd(dx/boxl);
    dy -= boxl*rnd(dy/boxl);
    dz -= boxl*rnd(dz/boxl);

    d2 = dx*dx+dy*dy+dz*dz;
    rep_tol = sigma_rep2[itype][jtype]*tol;
    if( d2 <  rep_tol ) continue;
    d6 = d2*d2*d2;
    d12 = d6*d6;

    co1 = force_coeff_rep[itype][jtype]/d2*
      (2.0*sigma_rep12[itype][jtype]/d12+sigma_rep6[itype][jtype]/d6);

    fx = co1*dx;
    fy = co1*dy;
    fz = co1*dz;

    force[ibead].x += fx;
    force[ibead].y += fy;
    force[ibead].z += fz;

    force[jbead].x -= fx;
    force[jbead].y -= fy;
    force[jbead].z -= fz;

  }

}

void soft_sphere_angular_forces()
{

  using namespace std;

  char line[2048];

  int ibead,kbead;
  double dx,dy,dz,d,d8;
  double fx,fy,fz;
  double co1;

  for( int i=1; i<=nang; i++ ) {

      ibead = ibead_ang[i];
      kbead = kbead_ang[i];

      dx = unc_pos[kbead].x - unc_pos[ibead].x;
      dy = unc_pos[kbead].y - unc_pos[ibead].y;
      dz = unc_pos[kbead].z - unc_pos[ibead].z;

      // min images

      dx -= boxl*rnd(dx/boxl);
      dy -= boxl*rnd(dy/boxl);
      dz -= boxl*rnd(dz/boxl);

      d = sqrt(dx*dx+dy*dy+dz*dz);
      d8 = pow(d,8.0);

      co1 = f_ang_ss_coeff/d8;

      fx = co1*dx;
      fy = co1*dy;
      fz = co1*dz;

      force[ibead].x -= fx;
      force[ibead].y -= fy;
      force[ibead].z -= fz;

      force[kbead].x += fx;
      force[kbead].y += fy;
      force[kbead].z += fz;

  }

}

void fene_forces()
{

  using namespace std;


  int ibead, jbead;
  double dx, dy, dz, d, dev, dev2;
  double fx, fy, fz;
  double temp;

  char line[2048];

  for( int i=1; i<=nbnd; i++ ) {

    ibead = ibead_bnd[i];
    jbead = jbead_bnd[i];

    dx = unc_pos[jbead].x-unc_pos[ibead].x;
    dy = unc_pos[jbead].y-unc_pos[ibead].y;
    dz = unc_pos[jbead].z-unc_pos[ibead].z;

    dx -= boxl*rnd(dx/boxl);
    dy -= boxl*rnd(dy/boxl);
    dz -= boxl*rnd(dz/boxl);

    d = sqrt(dx*dx+dy*dy+dz*dz);
    dev = d - pdb_dist[i];
    dev2 = dev*dev;
    temp = -k_bnd*dev/d/(1.0-dev2/R0sq);

    fx = temp*dx;
    fy = temp*dy;
    fz = temp*dz;

    force[ibead].x -= fx;
    force[ibead].y -= fy;
    force[ibead].z -= fz;

    force[jbead].x += fx;
    force[jbead].y += fy;
    force[jbead].z += fz;

  }

}

void random_force() {

  using namespace std;

  double var;
  int problem;

  var = sqrt(2.0*T*zeta/h);

  for( int i=1; i<=nbead; i++ ) {
    force[i].x += var*generator.gasdev();
    force[i].y += var*generator.gasdev();
    force[i].z += var*generator.gasdev();

  }

}

/**********************
* Start GPU Functions *
**********************/

void vdw_energy_gpu()
{
  e_vdw_rr = 0.0;
  e_vdw_rr_att = 0.0;
  e_vdw_rr_rep = 0.0;

  vdw_energy_att_gpu();

  vdw_energy_rep_gpu();

  e_vdw_rr = e_vdw_rr_att + e_vdw_rr_rep;

  return;
}

void vdw_energy_att_gpu(){	
	int N = nil_att + 1;
	
	int size_int = N*sizeof(int);
	int size_double = N*sizeof(double);
	int size_double3 = (nbead + 1)*sizeof(double3);

	int *dev_ibead_pair_list_att;
  int *dev_jbead_pair_list_att;
  int *dev_itype_pair_list_att;
  int *dev_jtype_pair_list_att;
	double *dev_pl_lj_nat_pdb_dist6;
	double *dev_pl_lj_nat_pdb_dist12;
	
	double3 *dev_unc_pos;
	
	double *dev_result;
	
	hipMalloc((void **)&dev_ibead_pair_list_att, size_int);
	hipMalloc((void **)&dev_jbead_pair_list_att, size_int);
	hipMalloc((void **)&dev_itype_pair_list_att, size_int);
	hipMalloc((void **)&dev_jtype_pair_list_att, size_int);
	hipMalloc((void **)&dev_pl_lj_nat_pdb_dist6, size_double);
	hipMalloc((void **)&dev_pl_lj_nat_pdb_dist12, size_double);
	
	hipMalloc((void **)&dev_unc_pos, size_double3);
	
	hipMalloc((void **)&dev_result, size_double);
	
	hipMemcpy(dev_ibead_pair_list_att, ibead_pair_list_att, size_int, hipMemcpyHostToDevice);
	hipMemcpy(dev_jbead_pair_list_att, jbead_pair_list_att, size_int, hipMemcpyHostToDevice);
	hipMemcpy(dev_itype_pair_list_att, itype_pair_list_att, size_int, hipMemcpyHostToDevice);
	hipMemcpy(dev_jtype_pair_list_att, jtype_pair_list_att, size_int, hipMemcpyHostToDevice);
	hipMemcpy(dev_pl_lj_nat_pdb_dist6, pl_lj_nat_pdb_dist6, size_double, hipMemcpyHostToDevice);
	hipMemcpy(dev_pl_lj_nat_pdb_dist12, pl_lj_nat_pdb_dist12, size_double, hipMemcpyHostToDevice);
	
	
	hipMemcpy(dev_unc_pos, unc_pos, size_double3, hipMemcpyHostToDevice);
	
	int threads = (int)min(N, SECTION_SIZE);
  int blocks = (int)ceil(1.0*N/SECTION_SIZE);
	
	vdw_energy_att_value_kernel<<<blocks, threads>>>(dev_ibead_pair_list_att, dev_jbead_pair_list_att, dev_itype_pair_list_att, dev_jtype_pair_list_att, 
														dev_pl_lj_nat_pdb_dist6, dev_pl_lj_nat_pdb_dist12, dev_unc_pos, N, boxl, dev_result);
	
	hier_ks_scan(dev_result, dev_result, N, 0);
	
	hipMemcpy(&e_vdw_rr_att, &dev_result[N-1], sizeof(double), hipMemcpyDeviceToHost);
	
	hipFree(dev_ibead_pair_list_att);
  hipFree(dev_jbead_pair_list_att);
  hipFree(dev_itype_pair_list_att);
  hipFree(dev_jtype_pair_list_att);
	hipFree(dev_pl_lj_nat_pdb_dist6);
	hipFree(dev_pl_lj_nat_pdb_dist12);
	
	hipFree(dev_unc_pos);
	
	hipFree(dev_result);
}

__global__ void vdw_energy_att_value_kernel(int *dev_ibead_pair_list_att, int *dev_jbead_pair_list_att, int *dev_itype_pair_list_att, int *dev_jtype_pair_list_att, 
											double *dev_pl_lj_nat_pdb_dist6, double *dev_pl_lj_nat_pdb_dist12, double3 *dev_unc_pos, int N, double boxl, double *dev_result){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i > 0 && i < N){
		int ibead,jbead;
		int itype,jtype;
		double dx,dy,dz,d,d2,d6,d12;
		
		ibead = dev_ibead_pair_list_att[i];
		jbead = dev_jbead_pair_list_att[i];
		itype = dev_itype_pair_list_att[i];
		jtype = dev_jtype_pair_list_att[i];

		dx = dev_unc_pos[jbead].x - dev_unc_pos[ibead].x;
		dy = dev_unc_pos[jbead].y - dev_unc_pos[ibead].y;
		dz = dev_unc_pos[jbead].z - dev_unc_pos[ibead].z;

		// min images

		// apply periodic boundary conditions to dx, dy, and dz
		//dx -= boxl*rnd(dx/boxl);
		double rnd_value;

		rnd_value = ( ((dx/boxl)>0) ? std::floor((dx/boxl)+0.5) : std::ceil((dx/boxl)-0.5) );
		dx -= boxl*rnd_value;

		//dy -= boxl*rnd(dy/boxl);
		rnd_value = ( ((dy/boxl)>0) ? std::floor((dy/boxl)+0.5) : std::ceil((dy/boxl)-0.5) );
		dy -= boxl*rnd_value;

		//dz -= boxl*rnd(dz/boxl);
		rnd_value = ( ((dz/boxl)>0) ? std::floor((dz/boxl)+0.5) : std::ceil((dz/boxl)-0.5) );
		dz -= boxl*rnd_value;

		d2 = dx*dx+dy*dy+dz*dz;
		d6 = d2*d2*d2;
		d12 = d6*d6;

		dev_result[i] = dev_coeff_att[itype][jtype] * (dev_pl_lj_nat_pdb_dist12[i]/d12)-2.0*(dev_pl_lj_nat_pdb_dist6[i]/d6);
	}else if(i == 0){
		dev_result[i] = 0;
	}
}

void vdw_energy_rep_gpu(){
	e_vdw_rr_rep = 0.0;
	
	int N = nil_rep + 1;
	
	int size_int = N*sizeof(int);
	int size_double = N*sizeof(double);
	int size_double3 = (nbead + 1)*sizeof(double3);

	int *dev_ibead_pair_list_rep;
  int *dev_jbead_pair_list_rep;
  int *dev_itype_pair_list_rep;
  int *dev_jtype_pair_list_rep;
	
	double3 *dev_unc_pos;
	
	double *dev_result;
	
	hipMalloc((void **)&dev_ibead_pair_list_rep, size_int);
	hipMalloc((void **)&dev_jbead_pair_list_rep, size_int);
	hipMalloc((void **)&dev_itype_pair_list_rep, size_int);
	hipMalloc((void **)&dev_jtype_pair_list_rep, size_int);
	
	hipMalloc((void **)&dev_unc_pos, size_double3);
	
	hipMalloc((void **)&dev_result, size_double);
	
	hipMemcpy(dev_ibead_pair_list_rep, ibead_pair_list_rep, size_int, hipMemcpyHostToDevice);
	hipMemcpy(dev_jbead_pair_list_rep, jbead_pair_list_rep, size_int, hipMemcpyHostToDevice);
	hipMemcpy(dev_itype_pair_list_rep, itype_pair_list_rep, size_int, hipMemcpyHostToDevice);
	hipMemcpy(dev_jtype_pair_list_rep, jtype_pair_list_rep, size_int, hipMemcpyHostToDevice);
	
	
	hipMemcpy(dev_unc_pos, unc_pos, size_double3, hipMemcpyHostToDevice);
	
	int threads = (int)min(N, SECTION_SIZE);
  int blocks = (int)ceil(1.0*N/SECTION_SIZE);
	
	vdw_energy_rep_value_kernel<<<blocks, threads>>>(dev_ibead_pair_list_rep, dev_jbead_pair_list_rep, dev_itype_pair_list_rep, dev_jtype_pair_list_rep, 
													                        dev_unc_pos, N, boxl, dev_result);
	
	hier_ks_scan(dev_result, dev_result, N, 0);
	
	hipMemcpy(&e_vdw_rr_rep, &dev_result[N-1], sizeof(double), hipMemcpyDeviceToHost);
	
	hipFree(dev_ibead_pair_list_rep);
  hipFree(dev_jbead_pair_list_rep);
  hipFree(dev_itype_pair_list_rep);
  hipFree(dev_jtype_pair_list_rep);
	
	hipFree(dev_unc_pos);
	
	hipFree(dev_result);
}

__global__ void vdw_energy_rep_value_kernel(int *dev_ibead_pair_list_rep, int *dev_jbead_pair_list_rep, int *dev_itype_pair_list_rep, int *dev_jtype_pair_list_rep, 
											double3 *dev_unc_pos, int N, double boxl, double *dev_result){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i > 0 && i < N){
		int ibead,jbead;
		int itype,jtype;
		double dx,dy,dz,d,d2,d6,d12;
		
		ibead = dev_ibead_pair_list_rep[i];
		jbead = dev_jbead_pair_list_rep[i];
		itype = dev_itype_pair_list_rep[i];
		jtype = dev_jtype_pair_list_rep[i];

		dx = dev_unc_pos[jbead].x - dev_unc_pos[ibead].x;
		dy = dev_unc_pos[jbead].y - dev_unc_pos[ibead].y;
		dz = dev_unc_pos[jbead].z - dev_unc_pos[ibead].z;

		// min images

		// apply periodic boundary conditions to dx, dy, and dz
		//dx -= boxl*rnd(dx/boxl);
		double rnd_value;

		rnd_value = ( ((dx/boxl)>0) ? std::floor((dx/boxl)+0.5) : std::ceil((dx/boxl)-0.5) );
		dx -= boxl*rnd_value;

		//dy -= boxl*rnd(dy/boxl);
		rnd_value = ( ((dy/boxl)>0) ? std::floor((dy/boxl)+0.5) : std::ceil((dy/boxl)-0.5) );
		dy -= boxl*rnd_value;

		//dz -= boxl*rnd(dz/boxl);
		rnd_value = ( ((dz/boxl)>0) ? std::floor((dz/boxl)+0.5) : std::ceil((dz/boxl)-0.5) );
		dz -= boxl*rnd_value;

		d2 = dx*dx+dy*dy+dz*dz;
		d6 = d2*d2*d2;
		d12 = d6*d6;

    double s = dev_sigma_rep[itype][jtype];
    double s2 = s*s;
    double s6 = s2*s2*s2;
    double s12 = s6*s6;
		
    dev_result[i] = dev_coeff_rep[itype][jtype] * (s12/d12 + s6/d6);
	}else if(i == 0){
		dev_result[i] = 0;
	}
}

void hier_ks_scan(double *dev_X, double *dev_Y, int N, int re){
    if(N <= SECTION_SIZE){
        ksScanInc<<<1, N>>>(dev_X, dev_Y, N);

        hipDeviceSynchronize();

        return;
    }else{
        int threads = (int)min(N, SECTION_SIZE);
        int blocks = (int)ceil(1.0*N/SECTION_SIZE);

        double *dev_S;
        hipMalloc((void**)&dev_S, (int)ceil(1.0*N/SECTION_SIZE) * sizeof(double));
        
        ksScanAuxInc<<<blocks, threads>>>(dev_X, dev_Y, N, dev_S);
        hipDeviceSynchronize();

        hier_ks_scan(dev_S, dev_S, (int)ceil(1.0*N/SECTION_SIZE), 1);
        hipDeviceSynchronize();
        
        sumIt<<<blocks, threads>>>(dev_Y, dev_S, N);
        hipDeviceSynchronize();

        hipFree(dev_S);

        return;
    }
}

__global__ void ksScanAuxExc (double *X, double *Y, int InputSize, double *S) {
    double val;
    
    __shared__ double XY[SECTION_SIZE];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < InputSize && threadIdx.x != 0){
        XY[threadIdx.x] = X[i-1];
    }else{
        XY[threadIdx.x] = 0;
    }

    for(unsigned int stride = 1; stride < blockDim.x; stride *=2){
        __syncthreads();
        if(threadIdx.x >= stride){
            val = XY[threadIdx.x - stride];
        }
        __syncthreads();
        if(threadIdx.x >= stride){
            XY[threadIdx.x] += val;
        }
    }

    __syncthreads();
    if(i < InputSize){
        Y[i] = XY[threadIdx.x];
    }
    
    __syncthreads();
    if(threadIdx.x == 0){
        S[blockIdx.x] = XY[SECTION_SIZE-1];
    }
}

__global__ void ksScanAuxInc (double *X, double *Y, int InputSize, double *S) {
    double val;
    
    __shared__ double XY[SECTION_SIZE];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < InputSize){
        XY[threadIdx.x] = X[i];
    }

    for(unsigned int stride = 1; stride < blockDim.x; stride *=2){
        __syncthreads();
        if(threadIdx.x >= stride){
            val = XY[threadIdx.x - stride];
        }
        __syncthreads();
        if(threadIdx.x >= stride){
            XY[threadIdx.x] += val;
        }
    }

    __syncthreads();
    if(i < InputSize){
        Y[i] = XY[threadIdx.x];
    }
    
    __syncthreads();
    if(threadIdx.x == 0){
        S[blockIdx.x] = XY[SECTION_SIZE-1];
    }
}

__global__ void ksScanExc (double *X, double *Y, double InputSize) {
    double val;
    
    __shared__ double XY[SECTION_SIZE];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(i < InputSize && threadIdx.x != 0){
        XY[threadIdx.x] = X[i-1];
    }else{
        XY[threadIdx.x] = 0;
    }

    for(unsigned int stride = 1; stride < blockDim.x; stride *=2){
        __syncthreads();
        if(threadIdx.x >= stride){
            val = XY[threadIdx.x - stride];
        }
        __syncthreads();
        if(threadIdx.x >= stride){
            XY[threadIdx.x] += val;
        }
    }

    __syncthreads();
    if(i < InputSize){
        Y[i] = XY[threadIdx.x];
    }
}

__global__ void ksScanInc (double *X, double *Y, int InputSize) {
    double val;
    
    __shared__ double XY[SECTION_SIZE];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(i < InputSize){
        XY[threadIdx.x] = X[i];
    }

    for(unsigned int stride = 1; stride < blockDim.x; stride *=2){
        __syncthreads();
        if(threadIdx.x >= stride){
            val = XY[threadIdx.x - stride];
        }
        __syncthreads();
        if(threadIdx.x >= stride){
            XY[threadIdx.x] += val;
        }
    }

    __syncthreads();
    if(i < InputSize){
        Y[i] = XY[threadIdx.x];
    }
}

__global__ void sumIt (double *Y, double *S, int InputSize) {
    if(blockIdx.x > 0){
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if(i < InputSize){
            Y[i] += S[blockIdx.x-1];
        }
    }
}

void vdw_forces_gpu()
{
  using namespace std;
  
  vdw_forces_att_gpu();

  vdw_forces_rep_gpu();
}

void vdw_forces_att_gpu(){
	int *dev_ibead_pair_list_att;
	int *dev_jbead_pair_list_att;
	int *dev_itype_pair_list_att;
	int *dev_jtype_pair_list_att;
	double *dev_pl_lj_nat_pdb_dist;
	double3 *dev_unc_pos;
	double3 *dev_force;
	
	int N = nil_att + 1;
	
	int size_int = N*sizeof(int);
	int size_double = N*sizeof(double);
	int size_double3 = N*sizeof(double3);
	
	hipMalloc((void **)&dev_ibead_pair_list_att, size_int);
	hipMalloc((void **)&dev_jbead_pair_list_att, size_int);
	hipMalloc((void **)&dev_itype_pair_list_att, size_int);
	hipMalloc((void **)&dev_jtype_pair_list_att, size_int);
	hipMalloc((void **)&dev_pl_lj_nat_pdb_dist, size_double);
	hipMalloc((void **)&dev_unc_pos, size_double3);
	hipMalloc((void **)&dev_force, size_double3);
	
	hipMemcpy(dev_ibead_pair_list_att, ibead_pair_list_att, size_int, hipMemcpyHostToDevice);
	hipMemcpy(dev_jbead_pair_list_att, jbead_pair_list_att, size_int, hipMemcpyHostToDevice);
	hipMemcpy(dev_itype_pair_list_att, itype_pair_list_att, size_int, hipMemcpyHostToDevice);
	hipMemcpy(dev_jtype_pair_list_att, jtype_pair_list_att, size_int, hipMemcpyHostToDevice);
	hipMemcpy(dev_pl_lj_nat_pdb_dist, pl_lj_nat_pdb_dist, size_double, hipMemcpyHostToDevice);
	hipMemcpy(dev_unc_pos, unc_pos, size_double3, hipMemcpyHostToDevice);
	hipMemcpy(dev_force, force, size_double3, hipMemcpyHostToDevice);
	
	int threads = (int)min(N, SECTION_SIZE);
	int blocks = (int)ceil(1.0*N/SECTION_SIZE);
	
	vdw_forces_att_kernel<<<blocks, threads>>>(dev_ibead_pair_list_att, dev_jbead_pair_list_att, dev_itype_pair_list_att, dev_jtype_pair_list_att, 
												dev_pl_lj_nat_pdb_dist, boxl, N, dev_unc_pos, dev_force);
												
	hipMemcpy(force, dev_force, size_double3, hipMemcpyDeviceToHost);
										
	hipFree(dev_ibead_pair_list_att);
	hipFree(dev_jbead_pair_list_att);
	hipFree(dev_itype_pair_list_att);
	hipFree(dev_jtype_pair_list_att);
	hipFree(dev_pl_lj_nat_pdb_dist);
	hipFree(dev_unc_pos);
	hipFree(dev_force);
}

__global__ void vdw_forces_att_kernel(int *dev_ibead_pair_list_att, int *dev_jbead_pair_list_att, int *dev_itype_pair_list_att, int *dev_jtype_pair_list_att, 
								double *dev_pl_lj_nat_pdb_dist, double boxl, int N, double3 *dev_unc_pos, double3 *dev_force){
									
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(i > 0 && i < N){
		int ibead,jbead;
		int itype,jtype;
		double dx,dy,dz,d,d2,d6,d12;
		double fx,fy,fz;
		double co1;
		const static double tol = 1.0e-7;

		ibead = dev_ibead_pair_list_att[i];
		jbead = dev_jbead_pair_list_att[i];
		itype = dev_itype_pair_list_att[i];
		jtype = dev_jtype_pair_list_att[i];

		dx = dev_unc_pos[jbead].x - dev_unc_pos[ibead].x;
		dy = dev_unc_pos[jbead].y - dev_unc_pos[ibead].y;
		dz = dev_unc_pos[jbead].z - dev_unc_pos[ibead].z;

		// apply periodic boundary conditions to dx, dy, and dz
		//dx -= boxl*rnd(dx/boxl);
		double rnd_value;

		rnd_value = ( ((dx/boxl)>0) ? std::floor((dx/boxl)+0.5) : std::ceil((dx/boxl)-0.5) );
		dx -= boxl*rnd_value;

		//dy -= boxl*rnd(dy/boxl);
		rnd_value = ( ((dy/boxl)>0) ? std::floor((dy/boxl)+0.5) : std::ceil((dy/boxl)-0.5) );
		dy -= boxl*rnd_value;

		//dz -= boxl*rnd(dz/boxl);
		rnd_value = ( ((dz/boxl)>0) ? std::floor((dz/boxl)+0.5) : std::ceil((dz/boxl)-0.5) );
		dz -= boxl*rnd_value;

		// compute square of distance between ibead and jbead
		d2 = dx*dx+dy*dy+dz*dz;
		
		double pl_dist2 = dev_pl_lj_nat_pdb_dist[i] * dev_pl_lj_nat_pdb_dist[i];
		
		if( d2 < tol*pl_dist2 ) return;
		d6 = d2*d2*d2;
		d12 = d6*d6;

		double pl_dist6 = pl_dist2 * pl_dist2 * pl_dist2;

		double pl_dist12 = pl_dist6 * pl_dist6;

		co1 = dev_force_coeff_att[itype][jtype]/d2*((pl_dist12/d12)-(pl_dist6/d6));

		fx = co1*dx;
		fy = co1*dy;
		fz = co1*dz;

		//dev_force[ibead].x += fx;
		atomicAdd(&dev_force[ibead].x, fx);
		
		//dev_force[ibead].y += fy;
		atomicAdd(&dev_force[ibead].y, fy);
		
		//dev_force[ibead].z += fz;
		atomicAdd(&dev_force[ibead].z, fz);

		//dev_force[jbead].x -= fx;
		atomicAdd(&dev_force[jbead].x, -1.0*fx);
		
		//dev_force[jbead].y -= fy;
		atomicAdd(&dev_force[jbead].y, -1.0*fy);
		
		//dev_force[jbead].z -= fz;
		atomicAdd(&dev_force[jbead].z, -1.0*fz);
	}
}

void vdw_forces_rep_gpu(){
	int *dev_ibead_pair_list_rep;
	int *dev_jbead_pair_list_rep;
	int *dev_itype_pair_list_rep;
	int *dev_jtype_pair_list_rep;
	double3 *dev_unc_pos;
	double3 *dev_force;
	
	int N = nil_rep + 1;
	
	int size_int = N*sizeof(int);
	int size_double3 = N*sizeof(double3);
	
	hipMalloc((void **)&dev_ibead_pair_list_rep, size_int);
	hipMalloc((void **)&dev_jbead_pair_list_rep, size_int);
	hipMalloc((void **)&dev_itype_pair_list_rep, size_int);
	hipMalloc((void **)&dev_jtype_pair_list_rep, size_int);
	hipMalloc((void **)&dev_unc_pos, size_double3);
	hipMalloc((void **)&dev_force, size_double3);
	
	hipMemcpy(dev_ibead_pair_list_rep, ibead_pair_list_rep, size_int, hipMemcpyHostToDevice);
	hipMemcpy(dev_jbead_pair_list_rep, jbead_pair_list_rep, size_int, hipMemcpyHostToDevice);
	hipMemcpy(dev_itype_pair_list_rep, itype_pair_list_rep, size_int, hipMemcpyHostToDevice);
	hipMemcpy(dev_jtype_pair_list_rep, jtype_pair_list_rep, size_int, hipMemcpyHostToDevice);
	
	hipMemcpy(dev_unc_pos, unc_pos, size_double3, hipMemcpyHostToDevice);
	hipMemcpy(dev_force, force, size_double3, hipMemcpyHostToDevice);
	
	int threads = (int)min(N, SECTION_SIZE);
	int blocks = (int)ceil(1.0*N/SECTION_SIZE);
	
	vdw_forces_rep_kernel<<<blocks, threads>>>(dev_ibead_pair_list_rep, dev_jbead_pair_list_rep, dev_itype_pair_list_rep, dev_jtype_pair_list_rep, 
												boxl, N, dev_unc_pos, dev_force);
										
										
	hipMemcpy(force, dev_force, size_double3, hipMemcpyDeviceToHost);
	
	hipFree(dev_ibead_pair_list_rep);
	hipFree(dev_jbead_pair_list_rep);
	hipFree(dev_itype_pair_list_rep);
	hipFree(dev_jtype_pair_list_rep);
	hipFree(dev_unc_pos);
	hipFree(dev_force);
}

__global__ void vdw_forces_rep_kernel(int *dev_ibead_pair_list_rep, int *dev_jbead_pair_list_rep, int *dev_itype_pair_list_rep, int *dev_jtype_pair_list_rep, double boxl, int N,
								double3 *dev_unc_pos, double3 *dev_force){
									
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if(i > 0 && i < N) {
		int ibead,jbead;
		int itype,jtype;
		double dx,dy,dz,d,d2,d6,d12;
		double fx,fy,fz;
		double co1;
		const static double tol = 1.0e-7;
		double rep_tol;

		ibead = dev_ibead_pair_list_rep[i];
		jbead = dev_jbead_pair_list_rep[i];
		itype = dev_itype_pair_list_rep[i];
		jtype = dev_jtype_pair_list_rep[i];

		dx = dev_unc_pos[jbead].x - dev_unc_pos[ibead].x;
		dy = dev_unc_pos[jbead].y - dev_unc_pos[ibead].y;
		dz = dev_unc_pos[jbead].z - dev_unc_pos[ibead].z;

		// apply periodic boundary conditions to dx, dy, and dz
		//dx -= boxl*rnd(dx/boxl);
		double rnd_value;

		rnd_value = ( ((dx/boxl)>0) ? std::floor((dx/boxl)+0.5) : std::ceil((dx/boxl)-0.5) );
		dx -= boxl*rnd_value;

		//dy -= boxl*rnd(dy/boxl);
		rnd_value = ( ((dy/boxl)>0) ? std::floor((dy/boxl)+0.5) : std::ceil((dy/boxl)-0.5) );
		dy -= boxl*rnd_value;

		//dz -= boxl*rnd(dz/boxl);
		rnd_value = ( ((dz/boxl)>0) ? std::floor((dz/boxl)+0.5) : std::ceil((dz/boxl)-0.5) );
		dz -= boxl*rnd_value;

		// compute square of distance between ibead and jbead
		d2 = dx*dx+dy*dy+dz*dz;

		double s2 = dev_sigma_rep[itype][jtype] * dev_sigma_rep[itype][jtype];
			
		rep_tol = s2*tol;

		if( d2 <  rep_tol ) return;
		d6 = d2*d2*d2;
		d12 = d6*d6;

		double s6 = s2*s2*s2;
		double s12 = s6*s6;

		co1 = dev_force_coeff_rep[itype][jtype]/d2 * (2.0*s12/d12+s6/d6);

		fx = co1*dx;
		fy = co1*dy;
		fz = co1*dz;

		//dev_force[ibead].x += fx;
		atomicAdd(&dev_force[ibead].x, fx);
		
		//dev_force[ibead].y += fy;
		atomicAdd(&dev_force[ibead].y, fy);
		
		//dev_force[ibead].z += fz;
		atomicAdd(&dev_force[ibead].z, fz);

		//dev_force[jbead].x -= fx;
		atomicAdd(&dev_force[jbead].x, -1.0*fx);
		
		//dev_force[jbead].y -= fy;
		atomicAdd(&dev_force[jbead].y, -1.0*fy);
		
		//dev_force[jbead].z -= fz;
		atomicAdd(&dev_force[jbead].z, -1.0*fz);

	}
}