#include "hip/hip_runtime.h"
#include <math.h>
#include <cstdlib>
#include "neighbor_list.h"
#include "global.h"
#include <stdio.h>
#include <stdlib.h>

#define SECTION_SIZE 1024

void update_neighbor_list() {

  double dx, dy, dz;
  double d2;
  int ibead, jbead, itype, jtype;
  double rcut, rcut2;

  nnl_att = 0;
  nnl_rep = 0;

  // calculations for native (attractiction) contacts
  for (int i=1; i<=ncon_att; i++) {
    // record sigma for ibead and jbead
    ibead = ibead_lj_nat[i];
    jbead = jbead_lj_nat[i];

    // record type of bead for ibead and jbead
    itype = itype_lj_nat[i];
    jtype = jtype_lj_nat[i];
    
    // calculate distance in x, y, and z for ibead and jbead
    dx = unc_pos[jbead].x - unc_pos[ibead].x;
    dy = unc_pos[jbead].y - unc_pos[ibead].y;
    dz = unc_pos[jbead].z - unc_pos[ibead].z;

    // apply periodic boundary conditions to dx, dy, and dz
    dx -= boxl*rnd(dx/boxl);
    dy -= boxl*rnd(dy/boxl);
    dz -= boxl*rnd(dz/boxl);

    // compute square of distance between ibead and jbead
    d2 = dx*dx+dy*dy+dz*dz;

    /* 
    Compute the cutoff distance for the given bead
    This is based off of lj_nat_pdb_dist[i], which is the distance 
    from ibead to jbead in the resulting folded structure
    */
    rcut = 3.2*lj_nat_pdb_dist[i];

    // square cutoff distance, since sqrt(d2) is computationally expensive
    rcut2 = rcut*rcut;

    // checks if distance squared is less than the cutoff distance squared
    if (d2 < rcut2) {
      // add to neighbor list
      nnl_att++;
      // add pair to respective attraction neighbor lists
      ibead_neighbor_list_att[nnl_att] = ibead;
      jbead_neighbor_list_att[nnl_att] = jbead;
      
      // record type of each bead
      itype_neighbor_list_att[nnl_att] = itype;
      jtype_neighbor_list_att[nnl_att] = jtype;

      // record values, so that calculatons are not repeated (look-up table)
      nl_lj_nat_pdb_dist[nnl_att] = lj_nat_pdb_dist[i];
      nl_lj_nat_pdb_dist2[nnl_att] = lj_nat_pdb_dist2[i];
      nl_lj_nat_pdb_dist6[nnl_att] = lj_nat_pdb_dist6[i];
      nl_lj_nat_pdb_dist12[nnl_att] = lj_nat_pdb_dist12[i];
    }
  }

  // calculations for non-native (repulsive) contacts
  for (int i=1; i<=ncon_rep; i++) {
    // record sigma for ibead and jbead
    ibead = ibead_lj_non_nat[i];
    jbead = jbead_lj_non_nat[i];

    // record type of bead for ibead and jbead
    itype = itype_lj_non_nat[i];
    jtype = jtype_lj_non_nat[i];

    // calculate distance in x, y, and z for ibead and jbead
    dx = unc_pos[jbead].x - unc_pos[ibead].x;
    dy = unc_pos[jbead].y - unc_pos[ibead].y;
    dz = unc_pos[jbead].z - unc_pos[ibead].z;

    // apply periodic boundary conditions to dx, dy, and dz
    dx -= boxl*rnd(dx/boxl);
    dy -= boxl*rnd(dy/boxl);
    dz -= boxl*rnd(dz/boxl);

    // compute square of distance between ibead and jbead
    d2 = dx*dx+dy*dy+dz*dz;

    /* 
    Compute the cutoff distance for the given bead
    This is based off of sigma_rep[itype][jtype],
    is based on the sigma for the types of ibead and jbead
    */
    rcut = 3.2*sigma_rep[itype][jtype];

    // square cutoff distance, since sqrt(d2) is computationally expensive
    rcut2 = rcut*rcut;

    // checks if distance squared is less than the cutoff distance squared
    if (d2 < rcut2) {
      // add to neighbor list
      nnl_rep++;

      // add pair to respective repulsive neighbor lists
      ibead_neighbor_list_rep[nnl_rep] = ibead;
      jbead_neighbor_list_rep[nnl_rep] = jbead;

      // record type of each bead
      itype_neighbor_list_rep[nnl_rep] = itype;
      jtype_neighbor_list_rep[nnl_rep] = jtype;
    }

  }
}

void update_neighbor_list_gpu() {
  int N;

  // Declare pointers for dev_output and dev_value arrays
  int *dev_value;
  int *dev_output;
  int *dev_unc_pos;

  // Start NL Update for Attractive Pairs
  int *dev_ibead_lj_nat;
  int *dev_jbead_lj_nat;
  int *dev_itype_lj_nat;
  int *dev_jtype_lj_nat;
  int *dev_lj_nat_pdb_dist;

  // Declare local variables

  // Calculate array size
  N = ncon_att+1;

  int size = (N) * sizeof(int);

  // Allocate dev_ arrays
  // TODO: Calculate correct size for each array
  hipMalloc((void**)&dev_value, size);
  hipMalloc((void**)&dev_output, size);
  hipMalloc((void**)&dev_ibead_lj_nat, size);
  hipMalloc((void**)&dev_jbead_lj_nat, size);
  hipMalloc((void**)&dev_itype_lj_nat, size);
  hipMalloc((void**)&dev_jtype_lj_nat, size);
  hipMalloc((void**)&dev_unc_pos, size);
  hipMalloc((void**)&dev_lj_nat_pdb_dist, size);

  // Copy arrays to dev_ arrays
  hipMemcpy(dev_ibead_lj_nat, ibead_lj_nat, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_jbead_lj_nat, jbead_lj_nat, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_itype_lj_nat, itype_lj_nat, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_jtype_lj_nat, jtype_lj_nat, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_unc_pos, unc_pos, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_lj_nat_pdb_dist, lj_nat_pdb_dist, size, hipMemcpyHostToDevice);

  int threads = (int)min(N, SECTION_SIZE);
  int blocks = (int)ceil(1.0*N/SECTION_SIZE);
  calculate_array_native<<<threads, blocks>>>(dev_ibead_lj_nat, dev_jbead_lj_nat, dev_itype_lj_nat, dev_jtype_lj_nat, dev_unc_pos, dev_lj_nat_pdb_dist, 
                                              dev_value, boxl, N);

  // Free memory used for calculating binary values
  hipFree(dev_unc_pos);
  

  // Run Scan on array
  hier_ks_scan(dev_value, dev_output, N, 0);
  int arrSize_att;
  int endVal_att;
  hipMemcpy(&arrSize_att, &dev_output[N-1], sizeof(int), hipMemcpyDeviceToHost); 
  hipMemcpy(&endVal_att, &dev_value[N-1], sizeof(int), hipMemcpyDeviceToHost); 

  // Increment arrSize by 1 if needed
  if(endVal_att){
      arrSize_att++;
  }

  // Declare more local variables
  int *dev_ibead_neighbor_list_att;
  int *dev_jbead_neighbor_list_att;
  int *dev_itype_neighbor_list_att;
  int *dev_jtype_neighbor_list_att;

  int *dev_lj_nat_pdb_dist2;
  int *dev_lj_nat_pdb_dist6;
  int *dev_lj_nat_pdb_dist12;

  // Allocate arrays
  hipMalloc((void**)&dev_ibead_neighbor_list_att, size);
  hipMalloc((void**)&dev_jbead_neighbor_list_att, size);
  hipMalloc((void**)&dev_itype_neighbor_list_att, size);
  hipMalloc((void**)&dev_jtype_neighbor_list_att, size);

  hipMalloc((void**)&dev_lj_nat_pdb_dist2, size);
  hipMalloc((void**)&dev_lj_nat_pdb_dist6, size);
  hipMalloc((void**)&dev_lj_nat_pdb_dist12, size);

  // Copy to from host to device
  hipMemcpy(dev_ibead_neighbor_list_att, ibead_neighbor_list_att, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_jbead_neighbor_list_att, jbead_neighbor_list_att, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_itype_neighbor_list_att, itype_neighbor_list_att, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_jtype_neighbor_list_att, jtype_neighbor_list_att, size, hipMemcpyHostToDevice);

  hipMemcpy(dev_lj_nat_pdb_dist2, lj_nat_pdb_dist2, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_lj_nat_pdb_dist6, lj_nat_pdb_dist6, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_lj_nat_pdb_dist12, lj_nat_pdb_dist12, size, hipMemcpyHostToDevice);

  // Copy elements
  threads = (int)min(N, SECTION_SIZE);
  blocks = (int)ceil(1.0*N/SECTION_SIZE);

  // Kernel to copy elements from dev_index to dev_output if their corresponding dev_value is 1
  copyElements<<<blocks, threads>>>(dev_ibead_lj_nat, dev_value, dev_output, dev_ibead_neighbor_list_att, N);
  copyElements<<<blocks, threads>>>(dev_jbead_lj_nat, dev_value, dev_output, dev_jbead_neighbor_list_att, N);
  copyElements<<<blocks, threads>>>(dev_jbead_lj_nat, dev_value, dev_output, dev_itype_neighbor_list_att, N);
  copyElements<<<blocks, threads>>>(dev_ibead_lj_nat, dev_value, dev_output, dev_jtype_neighbor_list_att, N);

  copyElements<<<blocks, threads>>>(dev_nl_lj_nat_pdb_dist, dev_value, dev_output, dev_lj_nat_pdb_dist, N);
  copyElements<<<blocks, threads>>>(dev_nl_lj_nat_pdb_dist2, dev_value, dev_output, dev_lj_nat_pdb_dist2, N);
  copyElements<<<blocks, threads>>>(dev_nl_lj_nat_pdb_dist6, dev_value, dev_output, dev_lj_nat_pdb_dist6, N);
  copyElements<<<blocks, threads>>>(dev_nl_lj_nat_pdb_dist12, dev_value, dev_output, dev_lj_nat_pdb_dist12, N);

  // Copy from device to host
  hipMemcpy(ibead_neighbor_list_att, dev_ibead_neighbor_list_att, arrSize_att*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(jbead_neighbor_list_att, dev_jbead_neighbor_list_att, arrSize_att*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(itype_neighbor_list_att, dev_itype_neighbor_list_att, arrSize_att*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(jtype_neighbor_list_att, dev_jtype_neighbor_list_att, arrSize_att*sizeof(int), hipMemcpyDeviceToHost);
  
  hipMemcpy(lj_nat_pdb_dist, dev_lj_nat_pdb_dist, arrSize_att*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(lj_nat_pdb_dist2, dev_lj_nat_pdb_dist2, arrSize_att*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(lj_nat_pdb_dist6, dev_lj_nat_pdb_dist6, arrSize_att*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(lj_nat_pdb_dist12, dev_lj_nat_pdb_dist12, arrSize_att*sizeof(int), hipMemcpyDeviceToHost);

  // Free the rest of the memory
  hipFree(dev_ibead_lj_nat);
  hipFree(dev_jbead_lj_nat);
  hipFree(dev_itype_lj_nat);
  hipFree(dev_jtype_lj_nat);
  hipFree(dev_lj_nat_pdb_dist);
  hipFree(dev_output);
  hipFree(dev_value);

  /*************************************************************************************************************************************/

  // Start NL Update for Repulsive Pairs
  int *dev_ibead_lj_non_nat;
  int *dev_jbead_lj_non_nat;
  int *dev_itype_lj_non_nat;
  int *dev_jtype_lj_non_nat;
  int *dev_lj_non_nat_pdb_dist;

  // Calculate array size
  N = ncon_rep+1;

  int size = (N) * sizeof(int);

  // Allocate dev_ arrays
  // TODO: Calculate correct size for each array
  hipMalloc((void**)&dev_value, size);
  hipMalloc((void**)&dev_output, size);
  hipMalloc((void**)&dev_ibead_lj_non_nat, size);
  hipMalloc((void**)&dev_jbead_lj_non_nat, size);
  hipMalloc((void**)&dev_itype_lj_non_nat, size);
  hipMalloc((void**)&dev_jtype_lj_non_nat, size);
  hipMalloc((void**)&dev_unc_pos, size);
  hipMalloc((void**)&dev_lj_nat_pdb_dist, size);

  // Copy arrays to dev_ arrays
  hipMemcpy(dev_ibead_lj_non_nat, ibead_lj_nat, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_jbead_lj_non_nat, jbead_lj_nat, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_itype_lj_non_nat, itype_lj_nat, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_jtype_lj_non_nat, jtype_lj_nat, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_unc_pos, unc_pos, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_lj_non_nat_pdb_dist, lj_nat_pdb_dist, size, hipMemcpyHostToDevice);

  threads = (int)min(N, SECTION_SIZE);
  blocks = (int)ceil(1.0*N/SECTION_SIZE);
  calculate_array_non_native<<<threads, blocks>>>(dev_ibead_lj_non_nat, dev_jbead_lj_non_nat, dev_itype_lj_non_nat, dev_jtype_lj_non_nat, dev_unc_pos,
                                                dev_lj_non_nat_pdb_dist, dev_value, boxl, N);

  // Free memory used for calculating binary values
  hipFree(dev_unc_pos);
  

  // Run Scan on array
  hier_ks_scan(dev_value, dev_output, N, 0);

  int arrSize_rep;
  int endVal_rep;
  hipMemcpy(&arrSize_rep, &dev_output[N-1], sizeof(int), hipMemcpyDeviceToHost); 
  hipMemcpy(&endVal_rep, &dev_value[N-1], sizeof(int), hipMemcpyDeviceToHost); 

  // Increment arrSize by 1 if needed
  if(endVal_rep){
      arrSize_rep++;
  }

  // Copy elements
  int threads = (int)min(N, SECTION_SIZE);
  int blocks = (int)ceil(1.0*N/SECTION_SIZE);

  // Kernel to copy elements from dev_index to dev_output if their corresponding dev_value is 1
  copyElements<<<blocks, threads>>>(dev_ibead_lj_non_nat, dev_value, dev_output, dev_ibead_neighbor_list_rep, N);
  copyElements<<<blocks, threads>>>(dev_jbead_lj_non_nat, dev_value, dev_output, dev_jbead_neighbor_list_rep, N);
  copyElements<<<blocks, threads>>>(dev_jbead_lj_non_nat, dev_value, dev_output, dev_itype_neighbor_list_rep, N);
  copyElements<<<blocks, threads>>>(dev_ibead_lj_non_nat, dev_value, dev_output, dev_jtype_neighbor_list_rep, N);

  hipFree(dev_value);
  hipFree(dev_output);

  // Copy from device to host
  hipMemcpy(ibead_neighbor_list_rep, dev_ibead_neighbor_list_rep, arrSize_rep*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(jbead_neighbor_list_rep, dev_jbead_neighbor_list_rep, arrSize_rep*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(itype_neighbor_list_rep, dev_itype_neighbor_list_rep, arrSize_rep*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(jtype_neighbor_list_rep, dev_jtype_neighbor_list_rep, arrSize_rep*sizeof(int), hipMemcpyDeviceToHost);

  // Free the rest of the memory
  hipFree(dev_ibead_lj_non_nat);
  hipFree(dev_jbead_lj_non_nat);
  hipFree(dev_itype_lj_non_nat);
  hipFree(dev_jtype_lj_non_nat);
  hipFree(dev_lj_non_nat_pdb_dist);
}

void calculate_array_native(int *&dev_ibead_lj_nat, int *&dev_jbead_lj_nat, int *&dev_itype_lj_nat, int *&dev_jtype_lj_nat, int *&dev_unc_pos, int *&dev_lj_nat_pdb_dist, 
                            int *&dev_value, int boxl, int N){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < N){
    double dx, dy, dz;
    double d2;
    int ibead, jbead, itype, jtype;
    double rcut, rcut2;

    // record sigma for ibead and jbead
    ibead = dev_ibead_lj_nat[i];
    jbead = dev_jbead_lj_nat[i];

    // record type of bead for ibead and jbead
    itype = dev_itype_lj_nat[i];
    jtype = dev_jtype_lj_nat[i];
    
    // calculate distance in x, y, and z for ibead and jbead
    dx = dev_unc_pos[jbead].x - dev_unc_pos[ibead].x;
    dy = dev_unc_pos[jbead].y - dev_unc_pos[ibead].y;
    dz = dev_unc_pos[jbead].z - dev_unc_pos[ibead].z;

    // apply periodic boundary conditions to dx, dy, and dz
    dx -= boxl*rnd(dx/boxl);
    dy -= boxl*rnd(dy/boxl);
    dz -= boxl*rnd(dz/boxl);

    // compute square of distance between ibead and jbead
    d2 = dx*dx+dy*dy+dz*dz;

    /* 
    Compute the cutoff distance for the given bead
    This is based off of lj_nat_pdb_dist[i], which is the distance 
    from ibead to jbead in the resulting folded structure
    */
    rcut = 3.2*sigma_rep[itype][jtype];

    // square cutoff distance, since sqrt(d2) is computationally expensive
    rcut2 = rcut*rcut;

    if(d2 < rcut2){
      dev_value[i] = 1;
    }else{
      dev_value[i] = 0;
    }
  }else if(i == 0){
    dev_value[0] = 0;
  }
}

void calculate_array_non_native(int *&dev_ibead_lj_non_nat, int *&dev_jbead_lj_non_nat, int *&dev_itype_lj_non_nat, int *&dev_jtype_lj_non_nat, 
                                int *&dev_unc_pos, int *&dev_lj_non_nat_pdb_dist, int *&dev_value, int boxl, int N){
  int i = blockIdx.x * blockDim.x + threadIdx.x+1;
  if(i < N){
    double dx, dy, dz;
    double d2;
    int ibead, jbead, itype, jtype;
    double rcut, rcut2;

    // record sigma for ibead and jbead
    ibead = dev_ibead_lj_nat[i];
    jbead = dev_jbead_lj_nat[i];

    // record type of bead for ibead and jbead
    itype = dev_itype_lj_nat[i];
    jtype = dev_jtype_lj_nat[i];
    
    // calculate distance in x, y, and z for ibead and jbead
    dx = dev_unc_pos[jbead].x - dev_unc_pos[ibead].x;
    dy = dev_unc_pos[jbead].y - dev_unc_pos[ibead].y;
    dz = dev_unc_pos[jbead].z - dev_unc_pos[ibead].z;

    // apply periodic boundary conditions to dx, dy, and dz
    dx -= boxl*rnd(dx/boxl);
    dy -= boxl*rnd(dy/boxl);
    dz -= boxl*rnd(dz/boxl);

    // compute square of distance between ibead and jbead
    d2 = dx*dx+dy*dy+dz*dz;

    /* 
    Compute the cutoff distance for the given bead
    This is based off of lj_nat_pdb_dist[i], which is the distance 
    from ibead to jbead in the resulting folded structure
    */
    rcut = 3.2*dev_lj_nat_pdb_dist[i];

    // square cutoff distance, since sqrt(d2) is computationally expensive
    rcut2 = rcut*rcut;

    if(d2 < rcut2){
      dev_value[i] = 1;
    }else{
      dev_value[i] = 0;
    }
  }
}

/*
 * Function: compact
 * -----------------
 *  Finds points in index with a 1 in value and stores them
 *
 *  index: array of indices to check
 *  value: binary value indicating if the corresponding index value is true (1) or false (0)
 *  N: number of elements in index and value
 *  result: pointer where compacted array is stored
 *
 *  Returns: arrSize, the size of the compacted array
 *           Note: result is modified in-place
 */

int compact(int *&dev_index, int *&dev_value, int N, int *&dev_result){
    // Perform hierarchical Kogge-Stone scan on dev_value array and store result in dev_output
    hier_ks_scan(dev_value, dev_output, N, 0);

    // Copy size of compacted array from device to host and store in arrSize
    /* 
     * TODO: If the entire array has 1 as the value, an exclusive scan will have N-1 as the last value in the array.
     * However, allocating an array with N-1 entries will not store all N values from the index array.
     * Change code to determine when we need to increment arrSize and when we don't.
     * Options include:
     *  1) Changing the hierarchical scan kernel to determine if the final value in the value array is 1
     *  2) Checking to see if the final value is 1 in the value array
     * Option 2 was selected, but please double-check this approach
     */ 
    int arrSize_att;
    int endVal_att;
    hipMemcpy(&arrSize_att, &dev_output[N-1], sizeof(int), hipMemcpyDeviceToHost); 
    hipMemcpy(&endVal_att, &dev_value[N-1], sizeof(int), hipMemcpyDeviceToHost); 

    // Increment arrSize by 1 if needed
    if(endVal_att){
        arrSize_att++;
    }

    // Declare and allocate dev_result array to store compacted indices on device (on GPU)
    int *dev_result;
    hipMalloc((void**)&dev_result, arrSize*sizeof(int));

    // Declare and allocate dev_index to store indecies (on GPU)
    int *dev_index;
    hipMalloc((void**)&dev_index, size);

    // Copy indices from host to device
    hipMemcpy(dev_index, index, size, hipMemcpyHostToDevice);

    /* Calculate number of threads and blocks to use for copying
     * If N < SECTION_SIZE (max # of threads per block), use N threads per block. Else, use SECTION_SIZE threads per block
     * Divides number of elements in array by SECTION_SIZE and rounds up, ensuring it uses the minimum number of blocks required
     */
    int threads = (int)min(N, SECTION_SIZE);
    int blocks = (int)ceil(1.0*N/SECTION_SIZE);

    // Kernel to copy elements from dev_index to dev_output if their corresponding dev_value is 1
    copyElements<<<blocks, threads>>>(dev_index, dev_value, dev_output, dev_result, N);
    
    // Sync device to ensure GPU computation is finished before proceeding
    hipDeviceSynchronize();

    // Allocate result array on host
    result = (int *)malloc(arrSize*sizeof(int));

    // Copy dev_result (compacted array of indices in GPU) to result array on host
    hipMemcpy(result, dev_result, arrSize*sizeof(int), hipMemcpyDeviceToHost); 
    
    // Free device memory
    hipFree(dev_result); 
    hipFree(dev_index);
    hipFree(dev_value);
    hipFree(dev_output);

    return arrSize;
}

/*
 * Function: copyElements
 * -----------------
 *  Copys values marked true (1) from index array to result array
 *
 *  dev_index: array of indices to check (on GPU)
 *  dev_value: binary value indicating if the corresponding dev_index value is true (1) or false (0) (on GPU)
 *  N: number of elements in dev_index and dev_value
 *  dev_result: pointer where compacted array is stored (on GPU)
 */

__global__ void copyElements(int *dev_index, int *dev_value, int *dev_output, int *dev_result, int N){
    int i = blockIdx.x * blockDim.x + threadIdx.x+1;
    if(dev_value[i] && i < N){
        dev_result[dev_output[i]-1] = dev_index[i];
    }
    return;
}

/*
 * Function: hier_ks_scan
 * -----------------
 *  
 *
 *  dev_index: array of indices to check (on GPU)
 *  dev_value: binary value indicating if the corresponding dev_index value is true (1) or false (0) (on GPU)
 *  N: number of elements in dev_index and dev_value
 *  dev_result: pointer where compacted array is stored (on GPU)
 */

void hier_ks_scan(int *dev_X, int *dev_Y, int N, int re){
    if(N <= SECTION_SIZE){
        ksScanInc<<<1, N>>>(dev_X, dev_Y, N);

        hipDeviceSynchronize();

        return;
    }else{
        int threads = (int)min(N, SECTION_SIZE);
        int blocks = (int)ceil(1.0*N/SECTION_SIZE);

        int *dev_S;
        hipMalloc((void**)&dev_S, (int)ceil(1.0*N/SECTION_SIZE) * sizeof(int));
        
        ksScanAuxInc<<<blocks, threads>>>(dev_X, dev_Y, N, dev_S);
        hipDeviceSynchronize();

        hier_ks_scan(dev_S, dev_S, (int)ceil(1.0*N/SECTION_SIZE), 1);
        hipDeviceSynchronize();
        
        sumIt<<<blocks, threads>>>(dev_Y, dev_S, N);
        hipDeviceSynchronize();

        hipFree(dev_S);

        return;
    }
}

__global__ void ksScanAuxExc (int *X, int *Y, int InputSize, int *S) {
    int val;
    
    __shared__ int XY[SECTION_SIZE];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < InputSize && threadIdx.x != 0){
        XY[threadIdx.x] = X[i-1];
    }else{
        XY[threadIdx.x] = 0;
    }

    for(unsigned int stride = 1; stride < blockDim.x; stride *=2){
        __syncthreads();
        if(threadIdx.x >= stride){
            val = XY[threadIdx.x - stride];
        }
        __syncthreads();
        if(threadIdx.x >= stride){
            XY[threadIdx.x] += val;
        }
    }

    __syncthreads();
    if(i < InputSize){
        Y[i] = XY[threadIdx.x];
    }
    
    __syncthreads();
    if(threadIdx.x == 0){
        S[blockIdx.x] = XY[SECTION_SIZE-1];
    }
}

__global__ void ksScanAuxInc (int *X, int *Y, int InputSize, int *S) {
    int val;
    
    __shared__ int XY[SECTION_SIZE];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < InputSize){
        XY[threadIdx.x] = X[i];
    }

    for(unsigned int stride = 1; stride < blockDim.x; stride *=2){
        __syncthreads();
        if(threadIdx.x >= stride){
            val = XY[threadIdx.x - stride];
        }
        __syncthreads();
        if(threadIdx.x >= stride){
            XY[threadIdx.x] += val;
        }
    }

    __syncthreads();
    if(i < InputSize){
        Y[i] = XY[threadIdx.x];
    }
    
    __syncthreads();
    if(threadIdx.x == 0){
        S[blockIdx.x] = XY[SECTION_SIZE-1];
    }
}

__global__ void ksScanExc (int *X, int *Y, int InputSize) {
    int val;
    
    __shared__ int XY[SECTION_SIZE];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(i < InputSize && threadIdx.x != 0){
        XY[threadIdx.x] = X[i-1];
    }else{
        XY[threadIdx.x] = 0;
    }

    for(unsigned int stride = 1; stride < blockDim.x; stride *=2){
        __syncthreads();
        if(threadIdx.x >= stride){
            val = XY[threadIdx.x - stride];
        }
        __syncthreads();
        if(threadIdx.x >= stride){
            XY[threadIdx.x] += val;
        }
    }

    __syncthreads();
    if(i < InputSize){
        Y[i] = XY[threadIdx.x];
    }
}

__global__ void ksScanInc (int *X, int *Y, int InputSize) {
    int val;
    
    __shared__ int XY[SECTION_SIZE];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(i < InputSize){
        XY[threadIdx.x] = X[i];
    }

    for(unsigned int stride = 1; stride < blockDim.x; stride *=2){
        __syncthreads();
        if(threadIdx.x >= stride){
            val = XY[threadIdx.x - stride];
        }
        __syncthreads();
        if(threadIdx.x >= stride){
            XY[threadIdx.x] += val;
        }
    }

    __syncthreads();
    if(i < InputSize){
        Y[i] = XY[threadIdx.x];
    }
}

__global__ void sumIt (int *Y, int *S, int InputSize) {
    if(blockIdx.x > 0){
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if(i < InputSize){
            Y[i] += S[blockIdx.x-1];
        }
    }
}

__global__ void init_kernel(){
 return;
}