
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void cudaCheck(){					
  hipError_t error = hipGetLastError();
  if(error != hipSuccess)
  {
    // print the CUDA error message and exit
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }else{
      printf("Success!\n");
      exit(0);
  }
}